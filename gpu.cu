#include "hip/hip_runtime.h"
#!/bin/bash

set -e

sed 1,8d $0 > /tmp/gpu.cu
nvcc --arch=sm_70 /tmp/gpu.cu -o gpu
./gpu

#include <iostream>
#include <cstdio>

int GetKBitOfN(int32_t n, int k) {
  return (n >> k) & 1;
}

int GetHighestSetBit(int32_t n) {
  return sizeof(int32_t) * 8 - __builtin_clz(n);
}

int32_t ExpModuloBit(int32_t power, int32_t modulo) {
  int32_t res = 1;
  int64_t curr_power = 10;

  for (int i = 0; i < GetHighestSetBit(power); ++i) {
    if (GetKBitOfN(power, i)) {
      res = (res * curr_power) % modulo;
    }
    curr_power = (curr_power * curr_power) % modulo;
  }

  return res;
}

const int N = 10000000;

__global__ void CalculateS(int32_t *res) {
  atomicAdd_system(res, 10);
  printf("%d\n", res);
}

int main() {
  int32_t *res;
  hipMallocManaged(&res, sizeof(res));
  *res = 0;

  CalculateS<<<1, 1>>>(res);

  std::cout << "N = " << N << std::endl;
  std::cout << *res << std::endl;
  return 0;
}
